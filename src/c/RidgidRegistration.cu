#include "hip/hip_runtime.h"
#include "RidgidRegistration.h"
#include "CudaImageBuffer.cuh"

struct  corrReport
{
	Vec<int> delta;
	double correlation;
	double staticSig;
	double overlapSig;
	unsigned int nVoxels;
};

void calcMaxROIs(const Overlap& overlap, Vec<int> imageExtents, comparedImages<unsigned int>& imStarts,
	comparedImages<unsigned int>& imSizes, Vec<int>& maxOverlapSize)
{
	comparedImages<int> localStarts;
	comparedImages<int> localSizes;
	localStarts.staticIm.x = std::numeric_limits<int>::max();
	localStarts.staticIm.y = std::numeric_limits<int>::max();
	localStarts.staticIm.z = std::numeric_limits<int>::max();
	localStarts.overlapIm.x = std::numeric_limits<int>::max();
	localStarts.overlapIm.y = std::numeric_limits<int>::max();
	localStarts.overlapIm.z = std::numeric_limits<int>::max();

	localSizes.staticIm.x = 0;
	localSizes.staticIm.y = 0;
	localSizes.staticIm.z = 0;
	localSizes.overlapIm.x = 0;
	localSizes.overlapIm.y = 0;
	localSizes.overlapIm.z = 0;

	for (int deltaX=overlap.deltaXmin; deltaX<overlap.deltaXmax; ++deltaX)
	{
		for (int deltaY=overlap.deltaYmin; deltaY<overlap.deltaYmax; ++deltaY)
		{
			for (int deltaZ=overlap.deltaZmin; deltaZ<overlap.deltaZmax; ++deltaZ)
			{
				comparedImages<int> mins;
				Vec<int> szs;

				mins.staticIm.x = std::max<int>(0,overlap.deltaXss+deltaX);
				mins.overlapIm.x = std::max<int>(0,-(overlap.deltaXss+deltaX));
				szs.x = std::min<int>(imageExtents.x,overlap.deltaXse+deltaX+1) - mins.staticIm.x;

				mins.staticIm.y = std::max<int>(0,overlap.deltaYss+deltaY);
				mins.overlapIm.y = std::max<int>(0,-(overlap.deltaYss+deltaY));
				szs.y = std::min<int>(imageExtents.y,overlap.deltaYse+deltaY+1) - mins.staticIm.y;

				mins.staticIm.z = std::max<int>(0,overlap.deltaZss+deltaZ);
				mins.overlapIm.z = std::max<int>(0,-(overlap.deltaZss+deltaZ));
				szs.z = std::min<int>(imageExtents.z,overlap.deltaZse+deltaZ+1) - mins.staticIm.z;

				localStarts.staticIm.x = std::min<int>(localStarts.staticIm.x,mins.staticIm.x);
				localStarts.staticIm.y = std::min<int>(localStarts.staticIm.y,mins.staticIm.y);
				localStarts.staticIm.z = std::min<int>(localStarts.staticIm.z,mins.staticIm.z);

				localSizes.staticIm.x = std::max<int>(localSizes.staticIm.x,mins.staticIm.x+szs.x);
				localSizes.staticIm.y = std::max<int>(localSizes.staticIm.y,mins.staticIm.y+szs.y);
				localSizes.staticIm.z = std::max<int>(localSizes.staticIm.z,mins.staticIm.z+szs.z);

				localStarts.overlapIm.x = std::min<int>(localStarts.overlapIm.x,mins.overlapIm.x);
				localStarts.overlapIm.y = std::min<int>(localStarts.overlapIm.y,mins.overlapIm.y);
				localStarts.overlapIm.z = std::min<int>(localStarts.overlapIm.z,mins.overlapIm.z);

				localSizes.overlapIm.x = std::max<int>(localSizes.overlapIm.x,mins.overlapIm.x+szs.x);
				localSizes.overlapIm.y = std::max<int>(localSizes.overlapIm.y,mins.overlapIm.y+szs.y);
				localSizes.overlapIm.z = std::max<int>(localSizes.overlapIm.z,mins.overlapIm.z+szs.z);

				maxOverlapSize.x = std::max<int>(maxOverlapSize.x,szs.x);
				maxOverlapSize.y = std::max<int>(maxOverlapSize.y,szs.y);
				maxOverlapSize.z = std::max<int>(maxOverlapSize.z,szs.z);
			}
		}
	}

	localSizes.staticIm.x -= localStarts.staticIm.x;
	localSizes.staticIm.y -= localStarts.staticIm.y;
	localSizes.staticIm.z -= localStarts.staticIm.z;
	localSizes.overlapIm.x -= localStarts.overlapIm.x;
	localSizes.overlapIm.y -= localStarts.overlapIm.y;
	localSizes.overlapIm.z -= localStarts.overlapIm.z;

	imStarts.staticIm.x = localStarts.staticIm.x;
	imStarts.staticIm.y = localStarts.staticIm.y;
	imStarts.staticIm.z = localStarts.staticIm.z;
	imStarts.overlapIm.x = localStarts.overlapIm.x;
	imStarts.overlapIm.y = localStarts.overlapIm.y;
	imStarts.overlapIm.z = localStarts.overlapIm.z;
	imSizes.staticIm.x = localSizes.staticIm.x;
	imSizes.staticIm.y = localSizes.staticIm.y;
	imSizes.staticIm.z = localSizes.staticIm.z;
	imSizes.overlapIm.x = localSizes.overlapIm.x;
	imSizes.overlapIm.y = localSizes.overlapIm.y;
	imSizes.overlapIm.z = localSizes.overlapIm.z;
}

void ridgidRegistration(const ImageContainer* staticImage, const ImageContainer* overlapImage, const Overlap& overlap,
	Vec<int>& bestDelta, double& maxCorrOut, unsigned int& bestN, int deviceNum, const char* fileName)
{
	bestDelta = Vec<int>(0,0,0);
	maxCorrOut = std::numeric_limits<double>::min();
	bestN = 0;

	if (staticImage==NULL || overlapImage==NULL)
		return;

	comparedImages<unsigned int> imStarts, imSizes;
	Vec<int> maxOverlapSize;
	Vec<int> staticImageExtents(staticImage->getWidth(),staticImage->getHeight(),staticImage->getDepth());

	calcMaxROIs(overlap,staticImageExtents,imStarts,imSizes,maxOverlapSize);
	if (imSizes.staticIm>=staticImage->getDims())
		fprintf(stderr,"Using Total Static Image Size!\n");
	if (imSizes.overlapIm>=overlapImage->getDims())
		fprintf(stderr,"Using Total Overlap Image Size!\n");

	time_t mainStart, mainEnd, mipStart, mipEnd, xStart, xEnd, zStart, zEnd;
	double mainSec=0, mipSec=0, xSec=0, xSecTotal=0.0, zSec=0, zSecTotal=0.0;
	Vec<int> deltaMins(overlap.deltaXmin,overlap.deltaYmin,overlap.deltaZmin);
	Vec<int> deltaMaxs(overlap.deltaXmax,overlap.deltaYmax,overlap.deltaZmax);

	time(&mainStart);

	unsigned int iterations = std::max<int>(1,(deltaMaxs.x-deltaMins.x)*(deltaMaxs.y-deltaMins.y));
	unsigned int curIter = 0;
	Vec<int> deltaSizes((deltaMaxs.x-deltaMins.x),(deltaMaxs.y-deltaMins.y),(deltaMaxs.z-deltaMins.z));

	comparedImages<unsigned int> starts;
	comparedImages<unsigned int> szs;

	printf("(%d) Deltas(%d to %d, %d to %d, %d to %d) Max Overlap(%d, %d, %d)",deviceNum,
		deltaMins.x,deltaMaxs.x,deltaMins.y,deltaMaxs.y,deltaMins.z,deltaMaxs.z,
		maxOverlapSize.x,maxOverlapSize.y,maxOverlapSize.z);

	//Start of MIPs registration
	{
		const float* staticMaxRoi = staticImage->getFloatConstROIData(imStarts.staticIm,imSizes.staticIm);
		const float* overlapMaxRoi = overlapImage->getFloatConstROIData(imStarts.overlapIm,imSizes.overlapIm);

		CudaImageBuffer<float> staticMaxRoiCuda(imSizes.staticIm,false,deviceNum);
		const size_t MAX_MEM = staticMaxRoiCuda.getGlobalMemoryAvailable();
		if (imSizes.staticIm.product()*imSizes.overlapIm.product()*2*sizeof(float)>MAX_MEM)
		{
			printf("Overlap Too Large!\n");
			return;
		}

		CudaImageBuffer<float> overlapMaxRoiCuda(imSizes.overlapIm,false,deviceNum);

		staticMaxRoiCuda.loadImage(staticMaxRoi,imSizes.staticIm);
		overlapMaxRoiCuda.loadImage(overlapMaxRoi,imSizes.overlapIm);

		staticMaxRoiCuda.maximumIntensityProjection();
		overlapMaxRoiCuda.maximumIntensityProjection();

		CudaImageBuffer<float> staticCudaIm(staticMaxRoiCuda.getDimension(),false,deviceNum);
		CudaImageBuffer<float> overlapCudaIm(overlapMaxRoiCuda.getDimension(),false,deviceNum);

		size_t memUsed = staticMaxRoiCuda.getMemoryUsed() + overlapMaxRoiCuda.getMemoryUsed() + staticCudaIm.getMemoryUsed() + overlapCudaIm.getMemoryUsed();

		printf(" Memory(%04.2f%%, %6.2fMB, %dMB)\n",(float)memUsed/MAX_MEM*100.0f,(float)memUsed/1024.0f/1024.0f,(int)(MAX_MEM/1024.0f/1024.0f));
		time(&mipStart);
		for (int deltaX=deltaMins.x; deltaX<deltaMaxs.x; ++deltaX)//, ++reportInd.x)
		{
			//reportInd.y = 0;
			time(&xStart);
			for (int deltaY=deltaMins.y; deltaY<deltaMaxs.y; ++deltaY)//, ++reportInd.y)
			{
				starts.staticIm.x = (unsigned int)std::max<int>(0,overlap.deltaXss+deltaX-imStarts.staticIm.x);
				starts.overlapIm.x = (unsigned int)std::max<int>(0,-(overlap.deltaXss+deltaX)-imStarts.overlapIm.x);
				szs.staticIm.x = szs.overlapIm.x = (unsigned int)std::min<int>(imSizes.staticIm.x,overlap.deltaXse+deltaX+1) - starts.staticIm.x;

				starts.staticIm.y = (unsigned int)std::max<int>(0,overlap.deltaYss+deltaY-imStarts.staticIm.y);
				starts.overlapIm.y = (unsigned int)std::max<int>(0,-(overlap.deltaYss+deltaY)-imStarts.overlapIm.y);
				szs.staticIm.y = szs.overlapIm.y = (unsigned int)std::min<int>(imSizes.staticIm.y,overlap.deltaYse+deltaY+1) - starts.staticIm.y;
				szs.staticIm.z = 1;
				szs.overlapIm.z = 1;

				staticCudaIm.copyROI(staticMaxRoiCuda,starts.staticIm,szs.staticIm);
				overlapCudaIm.copyROI(overlapMaxRoiCuda,starts.overlapIm,szs.overlapIm);

// 				float* overlapTemp = overlapCudaIm.retrieveImage();
// 				char buff[255];
// 				sprintf_s(buff,"overlap_x%03d_y%03d",deltaX,deltaY);
// 				writeImage(overlapTemp,szs.overlapIm,buff);
// 				delete[] overlapTemp;
// 
// 				float* staticTemp = staticCudaIm.retrieveImage();
// 				sprintf_s(buff,"static_%03d_y%03d",deltaX,deltaY);
// 				writeImage(staticTemp,szs.staticIm,buff);
// 				delete[] staticTemp;

				double curCorr = staticCudaIm.normalizeCovariance(&overlapCudaIm);

				if (curCorr>maxCorrOut)
				{
					maxCorrOut = curCorr;
					bestDelta.x = deltaX;
					bestDelta.y = deltaY;
					bestDelta.z = 0;
					bestN = szs.staticIm.x*szs.staticIm.y;
				}
				++curIter; 			
			}
			time(&xEnd);
			xSec = difftime(xEnd,xStart);
			xSecTotal += xSec;

#ifndef _DEBUG
			if (0==deltaX%20)
#else
			if (0==deltaX%10)
#endif // _DEBUG
			{
				printf("\t");
				for (int i=0; i<deviceNum; ++i)
					printf("  ");

				printf("(%d)  BestCorr:%6.4f(%4d,%4d,%3d)", deviceNum, maxCorrOut, bestDelta.x, bestDelta.y, bestDelta.z);
				printf(" Done:%5.2f%% deltaX= %+03d", (float)curIter/iterations*100.0, deltaX);
				double est = (iterations-curIter)*(xSecTotal/curIter);
				int estMin = (int)floor(est/60.0);
				int estSec = (int)floor(est)%60;
				printf(" X(sec):%4.1f avgY(sec):%5.3f Est(min):%d:%02d\n", xSec, xSec/deltaSizes.y, estMin, estSec);
			}
		}

		delete[] staticMaxRoi;
		delete[] overlapMaxRoi;
	}//End of MIPs registration

	time(&mipEnd);
	mipSec = difftime(mipEnd,mipStart);

	Vec<double> offsets;
	offsets.x = ((deltaMaxs.x-deltaMins.x)/3.0)/2.0;
	offsets.y = ((deltaMaxs.y-deltaMins.y)/3.0)/2.0;
	offsets.z = ((deltaMaxs.z-deltaMins.z)/3.0)/2.0;

	deltaMins.x = std::max<int>(overlap.deltaXmin, (int)round((double)bestDelta.x - offsets.x));
	deltaMaxs.x = std::min<int>(overlap.deltaXmax, (int)round((double)bestDelta.x + offsets.x));
	deltaMins.y = std::max<int>(overlap.deltaYmin, (int)round((double)bestDelta.y - offsets.y));
	deltaMaxs.y = std::min<int>(overlap.deltaYmax, (int)round((double)bestDelta.y + offsets.y));
	deltaMins.z = std::max<int>(overlap.deltaZmin, (int)round((double)bestDelta.z - offsets.z));
	deltaMaxs.z = std::min<int>(overlap.deltaZmax, (int)round((double)bestDelta.z + offsets.z));

	printf("   (%d) Delta (%d,%d,%d) max:%f mipsTotalTime(min):%d:%02d avgTime:%5.5f\n\n",deviceNum,bestDelta.x,bestDelta.y,bestDelta.z,maxCorrOut,floor(mipSec/60.0),(int)mipSec%60,mipSec/iterations);

	iterations = (deltaMaxs.z-deltaMins.z)*SQR(LOCAL_REGION*2+1);
	curIter = 0;
	maxCorrOut = -std::numeric_limits<double>::infinity();

	//Start of Z stack registration
	{
		const float* staticMaxRoi = staticImage->getFloatConstROIData(imStarts.staticIm,imSizes.staticIm);
		const float* overlapMaxRoi = overlapImage->getFloatConstROIData(imStarts.overlapIm,imSizes.overlapIm);

		CudaImageBuffer<float> staticMaxRoiCuda(imSizes.staticIm,false,deviceNum);
		CudaImageBuffer<float> overlapMaxRoiCuda(imSizes.overlapIm,false,deviceNum);

		staticMaxRoiCuda.loadImage(staticMaxRoi,imSizes.staticIm);
		overlapMaxRoiCuda.loadImage(overlapMaxRoi,imSizes.overlapIm);

		CudaImageBuffer<float> staticCudaIm(staticMaxRoiCuda.getDimension(),false,deviceNum);
		CudaImageBuffer<float> overlapCudaIm(overlapMaxRoiCuda.getDimension(),false,deviceNum);

		const size_t MAX_MEM = staticMaxRoiCuda.getGlobalMemoryAvailable();
		size_t memUsed = staticMaxRoiCuda.getMemoryUsed() + overlapMaxRoiCuda.getMemoryUsed() + staticCudaIm.getMemoryUsed() + overlapCudaIm.getMemoryUsed();

		printf(" Memory(%04.2f%%, %6.2fMB, %dMB)\n",(float)memUsed/MAX_MEM*100.0f,(float)memUsed/1024.0f/1024.0f,(int)(MAX_MEM/1024.0f/1024.0f));

		//reportInd.z = 0;
		for (int deltaZ=deltaMins.z; deltaZ<deltaMaxs.z; ++deltaZ)//, ++reportInd.z)
		{
			time(&zStart);
			//reportInd.x = bestDelta.x - deltaMins.x;
			for (int deltaX=bestDelta.x-LOCAL_REGION; deltaX<bestDelta.x+LOCAL_REGION; ++deltaX)//, ++reportInd.x)
			{
				//reportInd.y = bestDelta.y - deltaMins.y;
				for (int deltaY=bestDelta.y-LOCAL_REGION; deltaY<bestDelta.y+LOCAL_REGION; ++deltaY)//, ++reportInd.y)
				{
					starts.staticIm.x = (unsigned int)std::max<int>(0,overlap.deltaXss+deltaX-imStarts.staticIm.x);
					starts.overlapIm.x = (unsigned int)std::max<int>(0,-(overlap.deltaXss+deltaX)-imStarts.overlapIm.x);
					szs.staticIm.x = szs.overlapIm.x = (unsigned int)std::min<int>(imSizes.staticIm.x,overlap.deltaXse+deltaX+1) - starts.staticIm.x;

					starts.staticIm.y = (unsigned int)std::max<int>(0,overlap.deltaYss+deltaY-imStarts.staticIm.y);
					starts.overlapIm.y = (unsigned int)std::max<int>(0,-(overlap.deltaYss+deltaY)-imStarts.overlapIm.y);
					szs.staticIm.y = szs.overlapIm.y = (unsigned int)std::min<int>(imSizes.staticIm.y,overlap.deltaYse+deltaY+1) - starts.staticIm.y;

					starts.staticIm.z = (unsigned int)std::max<int>(0,overlap.deltaZss+deltaZ-imStarts.staticIm.z);
					starts.overlapIm.z = (unsigned int)std::max<int>(0,-(overlap.deltaZss+deltaZ)-imStarts.overlapIm.z);
					szs.staticIm.z = szs.overlapIm.z = (unsigned int)std::min<int>(imSizes.staticIm.z,overlap.deltaZse+deltaZ+1) - starts.staticIm.z;
					staticCudaIm.copyROI(staticMaxRoiCuda,starts.staticIm,szs.staticIm);
					overlapCudaIm.copyROI(overlapMaxRoiCuda,starts.overlapIm,szs.overlapIm);

// 					float* overlapTemp = overlapCudaIm.retrieveImage();
// 					char buff[255];
// 					sprintf_s(buff,"overlap_x%03d_y%03d_z%03d_z%s",deltaX,deltaY,deltaZ,"%04d");
// 					writeImage(overlapTemp,szs.overlapIm,buff);
// 					delete[] overlapTemp;
// 
// 					float* staticTemp = staticCudaIm.retrieveImage();
// 					sprintf_s(buff,"static_%03d_y%03d_z%03d_z%s",deltaX,deltaY,deltaZ,"%04d");
// 					writeImage(staticTemp,szs.staticIm,buff);
// 					delete[] staticTemp;

					float curCorr = staticCudaIm.normalizeCovariance(&overlapCudaIm);

					if (curCorr>maxCorrOut)
					{
						maxCorrOut = curCorr;
						bestDelta.z = deltaZ;
						bestN = (unsigned int)szs.staticIm.product();
					}
					++curIter;
				}
			}
			time(&zEnd);
			zSec = difftime(zEnd,zStart);
			zSecTotal += zSec;

#ifndef _DEBUG
			if (0==deltaZ%4)
#endif
			{
				printf("\t");
				for (int i=0; i<deviceNum; ++i)
					printf("  ");

				printf("(%d)  BestCorr:%6.4f(%4d,%4d,%3d)", deviceNum, maxCorrOut, bestDelta.x, bestDelta.y, bestDelta.z);
				printf(" Done:%5.2f%% deltaZ= %+02d", (float)curIter/iterations*100.0, deltaZ);
				double est = (iterations-curIter)*(zSecTotal/curIter);
				int estMin = (int)floor(est/60.0);
				int estSec = (int)floor(est)%60;
				printf(" Z(sec):%4.1f avgX(sec):%5.3f avgY(sec):%5.3f Est(min):%d:%02d\n", zSec, zSec/(2*LOCAL_REGION +1), zSecTotal/curIter, estMin, estSec);
			}
		}
		delete[] staticMaxRoi;
		delete[] overlapMaxRoi;
	}//End of Z stack registration

	time(&mainEnd);
	mainSec = difftime(mainEnd,mainStart);

	int totMin = (int)floor(mainSec/60.0);
	int totSec = (int)floor(mainSec)%60;
	printf("  (%d) Delta (%d,%d,%d) max:%5.5f totalTime(min):%d:%02d\n",deviceNum,bestDelta.x,bestDelta.y,bestDelta.z,maxCorrOut,totMin,totSec);
}
