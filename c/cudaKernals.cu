#include "hip/hip_runtime.h"
#include "cudaKernals.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaUtilities.h"

__global__ void meanFilter(float* imageIn, float* imageOut, int imageWidth, int imageHeight, int imageDepth, int kernalDiameter)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x<imageWidth && y<imageHeight && z<imageDepth)
	{
		int kernalRadius = kernalDiameter/2;
		float val = 0;
		int xMin = max(0,x-kernalRadius);
		int xMax = min(imageWidth,x+kernalRadius);
		int yMin = max(0,y-kernalRadius);
		int yMax = min(imageHeight,y+kernalRadius);
		int zMin = max(0,z-kernalRadius);
		int zMax = min(imageDepth,z+kernalRadius);

		for (int i=xMin; i<xMax; ++i)
		{
			for (int j=yMin; j<yMax; ++j)
			{
				for (int k=zMin; k<zMax; ++k)
					//center imageIn[x+y*imageWidth]
					val += imageIn[i+j*imageWidth+k*imageHeight*imageWidth];
			}
		}

		imageOut[x+y*imageWidth+z*imageHeight*imageWidth] = min(val/((xMax-xMin)*(yMax-yMin)*(zMax-zMin)),255.0f);
	}
}

__global__ void addConstantInPlace(float* image, int width, int height, int depth, int additive)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y	+ blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x<width && y<height && z<depth)
		image[x+y*width+z*height*width] = image[x+y*width+z*height*width] + additive;
}

__global__ void powerInPlace(float* image, int width, int height, int depth, int power)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y	+ blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x<width && y<height && z<depth)
		image[x+y*width+z*height*width] = pow(image[x+y*width+z*height*width],power);
}

__global__ void sqrtInPlace(float* image, int width, int height, int depth)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y	+ blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x<width && y<height && z<depth)
		image[x+y*width+z*height*width] = sqrt(image[x+y*width+z*height*width]);
}

__global__ void multiplyTwoImages(const float* imageIn1, const float* imageIn2, float* imageOut, int width, int height, int depth)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y	+ blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x<width && y<height && z<depth)
		imageOut[x+y*width+z*height*width] = imageIn1[x+y*width+z*height*width] * imageIn2[x+y*width+z*height*width];
}

__global__ void getROI(const float* imageIn, int orgSizeX, int orgSizeY, int orgSizeZ, float* imageOut, int startX, int startY, int startZ, int newWidth, int newHeight, int newDepth)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y	+ blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (   x>=startX && x<orgSizeX && x<startX+newWidth
		&& y>=startY && y<orgSizeY && y<startY+newHeight
		&& z>=startZ && z<orgSizeZ && z<startZ+newDepth)
	{
		unsigned int outIndex = (x-startX)+(y-startY)*newWidth+(z-startZ)*newHeight*newWidth;
		imageOut[outIndex] = imageIn[x+y*orgSizeX+z*orgSizeY*orgSizeX];
		//imageOut[outIndex] = x;
	}
}

__global__ void reduceArray(float* arrayIn, float* arrayOut, unsigned int n)
{
	//This algorithm was used from a this website:
	// http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
	// accessed 4/28/2013

	extern __shared__ float sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x*2 + tid;
	unsigned int gridSize = blockDim.x*2*gridDim.x;
	sdata[tid] = 0;

	while (i<n)
	{
		sdata[tid] = arrayIn[i];

		if (i+blockDim.x<n)
			sdata[tid] += arrayIn[i+blockDim.x];

		i += gridSize;
	}
	__syncthreads();


	if (blockDim.x >= 2048)
	{
		if (tid < 1024) 
			sdata[tid] += sdata[tid + 1024];
		__syncthreads();
	}
	if (blockDim.x >= 1024)
	{
		if (tid < 512) 
			sdata[tid] += sdata[tid + 512];
		__syncthreads();
	}
	if (blockDim.x >= 512)
	{
		if (tid < 256) 
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if (blockDim.x >= 256) {
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads(); 
	}
	if (blockDim.x >= 128) 
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads(); 
	}

	if (tid < 32) {
		if (blockDim.x >= 64) 
		{
			sdata[tid] += sdata[tid + 32];
			__syncthreads(); 
		}
		if (blockDim.x >= 32)
		{
			sdata[tid] += sdata[tid + 16];
			__syncthreads(); 
		}
		if (blockDim.x >= 16)
		{
			sdata[tid] += sdata[tid + 8];
			__syncthreads(); 
		}
		if (blockDim.x >= 8)
		{
			sdata[tid] += sdata[tid + 4];
			__syncthreads(); 
		}
		if (blockDim.x >= 4)
		{
			sdata[tid] += sdata[tid + 2];
			__syncthreads(); 
		}
		if (blockDim.x >= 2)
		{
			sdata[tid] += sdata[tid + 1];
			__syncthreads(); 
		}
	}

	if (tid==0)
		arrayOut[blockIdx.x] = sdata[0];
}

__global__ void reduceImage(float* imageIn, int width, int height, int depth, float* imageOut, int xNeighborhood, int yNeighborhood, int zNeighborhood)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y	+ blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x%xNeighborhood==0 && y%yNeighborhood==0 && z%zNeighborhood==0)
	{
		int newWidth = width/xNeighborhood;
		int newHeight = height/yNeighborhood;
		int newdepth = depth/zNeighborhood;
		int neighborhoodPixels = xNeighborhood*yNeighborhood*zNeighborhood;
		float val = 0.0f;

		for (int xInd=x; xInd<x+xNeighborhood&&xInd<width; ++xInd)
		{
			for (int yInd=y; yInd<y+yNeighborhood&&yInd<height; ++yInd)
			{
				for (int zInd=z; zInd<z+zNeighborhood&&zInd<depth; ++zInd)
				{
					val += imageIn[xInd+yInd*width+zInd*height*width];
				}
			}
		}

		imageOut[(x/xNeighborhood)+(y/yNeighborhood)*newWidth+(z/zNeighborhood)*newHeight*newWidth] = val/neighborhoodPixels;
	}
}